/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                     /* number of time steps */
    tpoints,                /* total points along string */
    rcode;                      /* generic return code */
float  values[MAXPOINTS+2],     /* values at time t */
       oldval[MAXPOINTS+2],     /* values at time (t-dt) */
       newval[MAXPOINTS+2];     /* values at time (t+dt) */


/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin(fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
// __device__ void do_math(int i)
// {
//    float dtime, c, dx, tau, sqtau;

//    dtime = 0.3;
//    c = 1.0;
//    dx = 1.0;
//    tau = (c * dtime / dx);
//    sqtau = tau * tau;
//    newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
// }

/**********************************************************************
*      Our Kernel
**********************************************************************/
__global__ void kernel(float *oldval, float *values, int tpoints, int nsteps)  {
    int idx = threadIdx.x;
    float old = oldval[idx+1], val = values[idx+1], newv = 0.0;
    float dtime = 0.3, c = 1.0, dx = 1.0, tau, sqtau;
    for(int i = 1; i <= nsteps; i++)  {
        /* global endpoints */
        if ((idx == 0) || (idx  == tpoints-1))
            newv = 0.0;
        else  {
            tau = (c * dtime / dx);
            sqtau = tau * tau;
            newv = (2.0 * val) - old + (sqtau *  (-2.0)*val);
        }
        
        /* Update old values with new values */
        old = val;
        val = newv;
    }
    values[idx+1] = val;
    __syncthreads();
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
  int numBlocks = 1;
  int threadBlocks = tpoints;
  float *oldval_d, *values_d;
  hipMalloc(&oldval_d, (tpoints+1)*sizeof(float));
  hipMemcpy(oldval_d, values, (tpoints+1)*sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&values_d, (tpoints+1)*sizeof(float));
  hipMemcpy(values_d, values, (tpoints+1)*sizeof(float), hipMemcpyHostToDevice);
//  cudaMalloc(&newval_d, (tpoints+1)*sizeof(float));
//  cudaMemcpy(newval_d, newval, (tpoints+1)*sizeof(float), cudaMemcpyHostToDevice);

  kernel<<<numBlocks, threadBlocks>>>(oldval_d, values_d, tpoints, nsteps);

  hipMemcpy(values, values_d, (tpoints+1)*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(oldval_d);
  hipFree(values_d);
//  cudaFree(newval_d);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *  Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
//    struct timespec st, end;
//    clock_gettime(CLOCK_REALTIME, &st);

    sscanf(argv[1],"%d",&tpoints);
    sscanf(argv[2],"%d",&nsteps);
    check_param();
    printf("Initializing points on the line...\n");
    init_line();
    printf("Updating all points for all time steps...\n");
    update();
    printf("Printing final results...\n");
    printfinal();
    printf("\nDone.\n\n");
    
//    clock_gettime(CLOCK_REALTIME, &end);
//    double secs = (( double)end.tv_sec - (double)st.tv_sec ) + (( double)end.tv_nsec - (double)st.tv_nsec )/1000000000.0;
//    printf("Time: %.6f seconds\n", secs);
    return 0;
}
